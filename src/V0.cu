#include "hip/hip_runtime.h"
#include "../include/vector.h"
#include "../include/bitonic.h"

#include <hip/hip_runtime.h>
#include <stdbool.h>

#define THREADS_PER_BLOCK 1024

__device__ void swap(int* arr, int i, int j, bool condition) {
    
    if(condition){
        int temp = arr[i];
        arr[i] = arr[j];
        arr[j] = temp;
    }
}

__global__ void bitonicExchange(int* arr, int threads, int stage, int step) {

    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < threads) {

        unsigned int partner = tid^step;
        if (partner > tid) {
            bool minmax = (tid & stage) == 0;
            swap(arr, tid, partner, minmax ? arr[tid] > arr[partner] : arr[tid] < arr[partner]);
        } else {
            tid += threads;
            partner += threads;

            bool minmax = (tid & stage) == 0;
            swap(arr, tid, partner, minmax ? arr[tid] < arr[partner] : arr[tid] > arr[partner]);
        }
    }
}

void bitonicSort(Vector v) {

    int n = v.n;
    int threads = n>>1;
    int blocks = (threads-1) / THREADS_PER_BLOCK+1;

    int* d_arr;
    int  size = n*sizeof(int);
    hipMalloc((void**)&d_arr, size);
    hipMemcpy(d_arr, v.arr, size, hipMemcpyHostToDevice);

    for (int stage=2; stage<=n; stage<<=1) {
        for (int step=stage>>1; step>0; step>>=1) {
            bitonicExchange<<<blocks, THREADS_PER_BLOCK>>>(d_arr, threads, stage, step);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(v.arr, d_arr, size, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}